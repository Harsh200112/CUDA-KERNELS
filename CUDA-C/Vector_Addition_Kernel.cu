#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectAdddition(float *A, float *B, float *C, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n){
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float *A, float *B, float *C, int n){
    float *A_d, *B_d, *C_d;
    size_t size = n * sizeof(float);
    hipMalloc((void**)&A_d, n);
    hipMalloc((void**)&B_d, n);
    hipMalloc((void**)&C_d, n);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(C_d, A, size, hipMemcpyHostToDevice);

    const unsigned int numThreads = 256;
    unsigned int numBlocks = (n + numThreads - 1)/numThreads;

    vectAdddition<<<numBlocks, numThreads>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(){
    const int n = 1000;
    
    float A[n], B[n], C[n];

    for (int i=0; i<n; i++){
        A[i] = float(i);
        B[i] = A[i] / 1000.0f;
    }

    vecAdd(A, B, C, n);

    for (int i=0; i<n; i++){
        printf("%8.3f", C[i]);
        printf(", ");
    }

    return 0;
}